#include "hip/hip_runtime.h"
/* All or part of this file was contributed by Intel under license:
 *   Copyright (C) 2017-2018 Intel Corporation
 *   SPDX-License-Identifier: MIT
 */

#include "tensors/gpu/add.h"

#include "tensors/gpu/cuda_helpers.h"

#include "functional/functional.h"
#include "functional/shape.h"
#include "functional/tensor.h"
#include "functional/tmp.h"

namespace marian {

namespace gpu {

template <size_t K, class Functor>
__global__ void gAddGeneric(Functor functor,
                            const functional::Shape full,
                            functional::Tensor<float> out,
                            functional::Array<functional::Tensor<float>, K> ins,
                            float scale = 1.0) {
  int outLength = out.shape().elements();
  bool same = outLength == full.elements();
  for(int i = 0; i < K; ++i)
    same = same && outLength == ins[i].shape().elements();

  constexpr size_t N = functional::Shape::size();
  functional::Array<int, N> len;
  for(int i = 0; i < N; ++i)
    len[i] = full[i] / out.shape()[i];

  functional::Array<int, N> dims;
  for(int bid = 0; bid < outLength; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < outLength) {
      if(same) {
        out[index] += functional::apply(functor, ins, index) * scale;
      } else {
        out.shape().dims(index, dims);
        out[index] += functional::loops(functor, ins, len, dims) * scale;
      }
    }
  }
}

template <size_t K, class Functor>
__global__ void gAddEqual(Functor functor,
                          functional::Tensor<float> out,
                          functional::Array<functional::Tensor<float>, K> ins,
                          float scale,
                          bool broadcast) {
  int length = out.shape().elements();
  functional::Array<int, functional::Shape::size()> dims;

  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      functional::Array<int, K> indices;
      indices.fill(index);

      if(broadcast) {
        out.shape().dims(index, dims);
        for(size_t i = 0; i < K; ++i)
          indices[i] = ins[i].shape().bindex(dims);
      }

      out[index] += functional::apply(functor, ins, indices) * scale;
    }
  }
}

template <size_t K, class Functor>
__global__ void gAddReduce(Functor functor,
                           const functional::Shape full,
                           functional::Tensor<float> out,
                           functional::Array<functional::Tensor<float>, K> ins,
                           float scale = 1.0) {
  int rows = full.elements() / full.back();
  int cols = full.back();

  bool same = true;
  for(int i = 0; i < K; ++i)
    same = same && ins[i].shape().elements() == full.elements();

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      if(same) {
        _sum[threadIdx.x] = 0;
        for(int tid = 0; tid < cols; tid += blockDim.x) {
          int id = tid + threadIdx.x;
          if(id < cols)
            _sum[threadIdx.x] += functional::apply(functor, ins, j * cols + id);
        }
      } else {
        functional::Array<int, functional::Shape::size()> dims;
        _sum[threadIdx.x] = 0;

        for(int tid = 0; tid < cols; tid += blockDim.x) {
          int id = tid + threadIdx.x;
          if(id < cols) {
            full.dims(j * cols + id, dims);
            functional::Array<int, K> indices;
            for(int i = 0; i < K; ++i)
              indices[i] = ins[i].shape().bindex(dims);
            _sum[threadIdx.x] += functional::apply(functor, ins, indices);
          }
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1)) {
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      out[j] += _sum[0] * scale;
    }
  }
}

template <class Functor, class... Tensors>
void Add(Functor functor, float scale, marian::Tensor out, Tensors... tensors) {
  hipSetDevice(out->getDevice().no);

  auto full = marian::Shape::broadcast({out, tensors...});

  int length = out->shape().elements();

  constexpr size_t K = sizeof...(Tensors);

  functional::Tensor<float> gOut = out;
  functional::Array<functional::Tensor<float>, K> gIns = {tensors...};

  if(full.back() != 1 && out->shape().back() == 1) {
    size_t m = full.elements() / length;
    size_t k = full.back();

    int blocks = std::min(MAX_BLOCKS, (int)m);
    int threads = std::min(MAX_THREADS, (int)k);
    int shared = sizeof(float) * threads * 2;

    gAddReduce<<<blocks, threads, shared>>>(functor, full, gOut, gIns, scale);

  } else if(out->shape() == full) {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    bool broadcast = false;
    for(int i = 0; i < K; ++i)
      broadcast = broadcast || gOut.shape() != gIns[i].shape();
    gAddEqual<<<blocks, threads>>>(functor, gOut, gIns, scale, broadcast);
  } else {
    int threads = std::min(MAX_THREADS, length);
    int blocks
        = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

    gAddGeneric<<<blocks, threads>>>(functor, full, gOut, gIns, scale);
  }
}

#include "tensors/gpu/add.inc"
}
}
